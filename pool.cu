#include "hip/hip_runtime.h"
/* Example of using lodepng to load, process, save image */
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 512

//Putting blocks of size width divided by 0, so that each thread can access the neighboring values. There is no neighboring value that is called twice.

__global__ void pool(unsigned char * d_out, unsigned char * d_in,int width,int height){

	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int i = (ind + width - 1) / width;
	int j = (ind + width - 1) % width;
	int k = (ind + 3) % 4;
	int size = width * height * 4;

	if(ind<2000) {
		printf("ind: %05d and width is %d height is %d : (%d,%d,%d)\n",ind,width,height,i,j,k);
	}
	if((j % 100 == 0 )&&(i % 100 == 0) && ind < size){
		printf("Original ind: %010d at (%04d,%04d,%04d)\n",ind,i,j,k);
	}


	//unsigned char max;
	// int new_width = (width+1)/2;
	// if(j%2 == 0 && k != 3){
		//max = d_in[4*width*i + 4*j + k];
		//if(d_in[4*width*(i+1) + 4*j + k]>max) max = d_in[4*width*(i+1) + 4*j + k];
		//if(d_in[4*width*(i+1) + 4*(j+1) + k]>max) max = d_in[4*width*(i+1) + 4*(j+1) + k];
		//if(d_in[4*width*i + 4*(j+1) + k]>max) max = d_in[4*width*i + 4*(j+1) + k];
		// d_out[new_width*i + j*2 + k] = max;
		// if(j < 10 )printf("Original max at (%d,%d,%d) for ind = %d\n",i,j,k,ind);
	// }
	// if(j % 2 == 0 && k == 3){
		//d_out[new_width * i + j*2 + 3] = d_in[4*width*i + 4*j + 3];
	// 	if(j < 10 )printf("Original max at (%d,%d,%d) for ind = %d\n",i,j,k,ind);
	// }
}


int process(char* input_filename, char* output_filename){
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	unsigned new_width, new_height;

	//image --> h_in
	//new_image --> h_out

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
		return error;
	}
	new_width = (width+1)/2;
	new_height = (height+1)/2;

	const int size = width * height * 4 * sizeof(unsigned char);
	const int new_size = new_width * new_height * 4 * sizeof(unsigned char);

	new_image = (unsigned char *)malloc(new_size);


	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, new_size);

	// transfer the array to the GPU
	hipMemcpy(d_in, image, size, hipMemcpyHostToDevice);

	printf("%d total size with width %d and height %d in %d blocks of size %d\n",size,width,height, (size+(BLOCK_WIDTH-1))/BLOCK_WIDTH, BLOCK_WIDTH);

	// launch the kernel
	dim3 dimGrid((size+(BLOCK_WIDTH-1))/BLOCK_WIDTH);
	dim3 dimBlock(BLOCK_WIDTH);


	pool<<<dimGrid, dimBlock>>>(d_out, d_in,width,height);

	// copy back the result array to the CPU
	hipMemcpy(new_image, d_out, new_size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	//lodepng_encode32_file(output_filename, new_image, width, height);
	//int i;
	//for(i = 0; i<128;i++)printf("new_image[%d] = %d\n",i,new_image[i]);

	free(image);
	free(new_image);
	return 0;
}

int main(int argc, char *argv[]){
	if ( argc >= 3 ){
		char* input_filename = argv[1];
		char* output_filename = argv[2];

		int error = process(input_filename, output_filename);

		if(error != 0){
			printf("An error occured. ( %d )\n",error);

		}else{
			printf("The rectification ran with success.\n");
		}
	}else{
		printf("There is inputs missing.\n");
	}
	return 0;
}
