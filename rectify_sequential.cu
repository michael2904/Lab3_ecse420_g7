#include "hip/hip_runtime.h"
/* Example of using lodepng to load, process, save image */
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 1000


__global__ void rectify(float * d_out, float * d_in){
	int idx = threadIdx.x;
	float f = d_in[idx];
	if(idx % 4 == 3){
		f = f < 127 ? 127 : f; // R
	}
	d_out[idx] = f;
}


void process(char* input_filename, char* output_filename)
{
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	const int size = width * height * 4 * sizeof(unsigned char);

	//image --> h_in
	//new_image --> h_out

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
	new_image = malloc(size);


	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, size);

	// transfer the array to the GPU
	hipMemcpy(d_in, image, size, hipMemcpyHostToDevice);

	// launch the kernel
	rectify<<<size/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(new_image, d_out, size, hipMemcpyDeviceToHost);

	// // rectify image
	// unsigned char value;
	// int i,j;
	// for (i = 0; i < height; i++) {
	// 	for (j = 0; j < width; j++) {
	// 		for (int k = 0; k < 3; k++) {
	// 			new_image[4*width*i + 4*j + k] = image[4*width*i + 4*j + k] < 127 ? 127 : image[4*width*i + 4*j + k]; // R
	// 		}
	// 		new_image[4*width*i + 4*j + 3] = image[4*width*i + 4*j + 3]; // A
	// 	}
	// }

	hipFree(d_in);
	hipFree(d_out);

	lodepng_encode32_file(output_filename, new_image, width, height);

	free(image);
	free(new_image);
}

int main(int argc, char *argv[])
{
	char* input_filename = argv[1];
	char* output_filename = argv[2];

	process(input_filename, output_filename);

	return 0;
}
