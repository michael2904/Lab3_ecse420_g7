#include "hip/hip_runtime.h"
/* Example of using lodepng to load, process, save image */
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 512

//Putting blocks of size width divided by 0, so that each thread can access the neighboring values. There is no neighboring value that is called twice.

__global__ void pool(int * d_out, unsigned char * d_in){
	int N = 998;
	int idx = threadIdx.x;
	int jdx = threadIdx.y;
	int kdx = threadIdx.z;
	int Bx = blockDim.x;
	int By = blockDim.y;
	int Bz = blockDim.z;
	int Bix = blockIdx.x;
	int Biy = blockIdx.y;
	int Biz = blockIdx.z;
	//int index = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = idx + jdx * N;
	if(index < 100){
		printf("Dimensions are Bx:%d By:%d Bz:%d Index: %05d indexes are: Bix:%d Biy:%d Biz:%d -- Threads are Tx:%d Ty:%d Tz: %d -- coord (%d,%d,%d) col:%d, row:%d\n", Bx,By,Bz,index,Bix,Biy,Biz,threadIdx.x,threadIdx.y, threadIdx.z,kdx,jdx,idx,col,row);
	}

	//unsigned char max;
	// int new_width = width/2;
	//     if(jdx%2 == 0 && kdx != 3){
	//         max = d_in[4*width*idx + 4*jdx + kdx];
	//         if(blockIdx.x == 0)printf("Original max = %d at (%d,%d,%d) for index = %d\n",max,idx,jdx,kdx,index);
	//         if(d_in[4*width*(idx+1) + 4*jdx + kdx]>max) max = d_in[4*width*(idx+1) + 4*jdx + kdx];
	//         if(d_in[4*width*(idx+1) + 4*(jdx+1) + kdx]>max) max = d_in[4*width*(idx+1) + 4*(jdx+1) + kdx];
	//         if(d_in[4*width*idx + 4*(jdx+1) + kdx]>max) max = d_in[4*width*idx + 4*(jdx+1) + kdx];
	//         d_out[new_width*idx + jdx*2 + kdx] = max;
	//         if(blockIdx.x == 0)printf("Not max = %d and stored %d at %d, at (%d,%d,%d) for index = %d\n",max,d_out[new_width*idx + jdx*2 + kdx],new_width*idx + jdx*2 + kdx,idx,jdx,kdx,index);
	//     }
	//     if(jdx % 2 == 0 && kdx == 3){
	//         d_out[new_width * idx + jdx*2 + 3] = d_in[4*width*idx + 4*jdx + 3];
	//     }
	//d_out[index] = index;
	//printf("Dimensions are Bx:%d By:%d Bz:%d Index: %05d indexes are: Bix:%d Biy:%d Biz:%d -- Threads are Tx:%d Ty:%d Tz: %d -- coord (%d,%d,%d)\n", Bx,By,Bz,index,Bix,Biy,Biz,threadIdx.x,threadIdx.y, threadIdx.z,kdx,jdx,idx);
	//printf("This is the index %d and this is d_out %d\n",index,d_out[index]);
}


int process(char* input_filename, char* output_filename){
	unsigned error;
	unsigned char *image, *new_image;
	unsigned width, height;
	unsigned new_width, new_height;

	//image --> h_in
	//new_image --> h_out

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
		return error;
	}
	new_width = (width+1)/2;
	new_height = (height+1)/2;

	const int size = width * height * 4 * sizeof(unsigned char);
	const int new_size = new_width * new_height * 4 * sizeof(int);

	const int block_quantity = (size+(BLOCK_WIDTH-1))/(BLOCK_WIDTH * 2 * 4);
	new_image = (unsigned char *)malloc(new_size);


	// declare GPU memory pointers
	unsigned char * d_in;
	int * d_out;

	// allocate GPU memory
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, new_size);

	// transfer the array to the GPU
	hipMemcpy(d_in, image, size, hipMemcpyHostToDevice);

	printf("%d total threads in %d blocks of size %d\n",size, block_quantity, BLOCK_WIDTH);

	// launch the kernel
	dim3 dimGrid((width+(BLOCK_WIDTH-1))/BLOCK_WIDTH, (height+(BLOCK_WIDTH-1))/BLOCK_WIDTH);
	dim3 dimBlock(BLOCK_WIDTH, 2);


	pool<<<dimGrid, dimBlock>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(new_image, d_out, new_size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	//lodepng_encode32_file(output_filename, new_image, width, height);
	int i;
	for(i = 0; i<128;i++)printf("new_image[%d] = %d\n",i,new_image[i]);

	free(image);
	free(new_image);
	return 0;
}

int main(int argc, char *argv[]){
	if ( argc >= 3 ){
		char* input_filename = argv[1];
		char* output_filename = argv[2];

		int error = process(input_filename, output_filename);

		if(error != 0){
			printf("An error occured. ( %d )\n",error);

		}else{
			printf("The rectification ran with success.\n");
		}
	}else{
		printf("There is inputs missing.\n");
	}
	return 0;
}
